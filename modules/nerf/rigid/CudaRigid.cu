#include "cuda/Types.h"
#include "cuda/CudaHelper.h"

#include "rigid/CudaRigid.cuh"
#include "rigid/CudaRigidGlob.cuh"

#include <iostream>

// #include "thrust/device_ptr.h"

#include <chrono>
#include <thread>

using namespace std::chrono_literals;

void CudaRigid::Initialize(const GLuint& vbo)
{
    cuda::registerGLBufferObject(vbo, &cudaVbo);
    cuda::computeGridSize(config.numParticles, 64, numBlocks, numThreads);

    hipMalloc((void **)&d_rngStates, numBlocks * numThreads * sizeof(hiprandState));
    rigid::initRNG<<<numBlocks, numThreads>>>(d_rngStates, 1000);

    // auto mapAndCall = [&cudaVbo](){
    //     float* dPos = (float *)cuda::mapGLBufferObject(&cudaVbo);
    //     initialize<<<numBlocks, numThreads>>>(dPos, config.numParticles, 
    //         d_rngStates, config.dataSize);
    //     cuda::unmapGLBufferObject(cudaVbo);
    // };

    float* dPos = (float *)cuda::mapGLBufferObject(&cudaVbo);
    rigid::initialize<<<numBlocks, numThreads>>>(dPos, config.numParticles, 
        d_rngStates, config.dataSize);
    cuda::unmapGLBufferObject(cudaVbo);
}

void CudaRigid::Call(const float dt) 
{
    float* dPos = (float *)cuda::mapGLBufferObject(&cudaVbo);
    rigid::update<<<numBlocks, numThreads>>>(dPos, config.numParticles, 
        dt, config.particleRaius, config.dataSize);

    GET_CUDA_ERROR("Kernel execution failed");

    cuda::unmapGLBufferObject(cudaVbo);
}