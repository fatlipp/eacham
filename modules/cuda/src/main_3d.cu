#include "hip/hip_runtime.h"
#include "particle.h"

#include <opencv2/opencv.hpp>
#include <opencv2/core.hpp>

#include <stdlib.h>
#include <stdio.h>
#include <vector>

__global__ void advanceParticles(unsigned char* inputImage)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;
	const int width = gridDim.x * blockDim.x;
	const int idx = (y * 4 + x) + blockIdx.z * (blockDim.x * blockDim.y * gridDim.x * gridDim.y);

	printf("GPU: xy: (%i, %i, %i), id: %i, data: %i\n", x, y, z, idx, 0);// inputImage[idx]);
}

int main(int argc, char ** argv)
{
	std::vector<unsigned char> data = {};

	for (int i = 0; i < 48; ++i)
		data.push_back(i);

	int totalBytes = data.size() * sizeof(unsigned char);

	unsigned char* inputImageArray = NULL;
	hipMalloc(&inputImageArray, totalBytes);
	hipDeviceSynchronize();

	auto error = hipGetLastError();
	
	if (error != hipSuccess)
  	{
        printf("1 %s\n",hipGetErrorString(error));
        exit(1);
  	}

	hipMemcpy(inputImageArray, data.data(), totalBytes, hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
	error = hipGetLastError();
	
	if (error != hipSuccess)
  	{
        printf("2 %s\n",hipGetErrorString(error));
        exit(1);
  	}

	dim3 dimBlock(2, 2);
	dim3 dimGrid(2, 2, 3);

	advanceParticles<<<dimGrid, dimBlock>>>(inputImageArray);
	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("3 %s\n",hipGetErrorString(error));
		exit(1);
	}

	hipDeviceSynchronize();
	hipMemcpy(data.data(), inputImageArray, totalBytes, hipMemcpyDeviceToHost);

	std::cout << "done\n";
	return 0;
}